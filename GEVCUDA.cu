
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 500 

// Declared  new variable type as array
typedef double arr[N];

/******** Code for elementwise matrix multiplication for dot product*********************/

__global__ void dotmat(arr *A, arr *Temp, arr *q, int k,arr *temp) {

    int row=blockIdx.x;
    int col=blockIdx.y;
    if( col<k){
        temp[row][col]=A[row][k]*q[row][col];
     }
}

/******summing up above matrix column wise to get k dot products******************/

__global__ void dot(arr *temp,arr *r,arr *Temp, arr *q, int k) {

    int j;
    int x=threadIdx.x;
    r[0][x]=0.0;
    if(x<k){
        for(j=0;j<N;j++){
          r[0][x]+=temp[j][x];
         }
      }
}

/**********suming over all dot products and respective column of Q**************/
__global__ void submat(arr *LR, arr *Temp, arr *q, int k,arr *r) {

    int row=blockIdx.x;
    int col=blockIdx.y;
     if( col<k){
        LR[row][col]=r[0][col]*q[row][col];
     }
}
/**********Add all the columns element wise into single vector and substract from A[k] and calculate q[k]********************/

__global__ void Qcal(arr *LR, arr *Temp, arr *q, int k) {

  __shared__ double p[N];
  __shared__ double nr;
    int  j;
    int x=threadIdx.x;
    p[x]=0.0;
    for(j=0;j<k;j++){
          p[x]+=LR[x][j];
    }
   __syncthreads();
    Temp[x][k]=Temp[x][k]-p[x];
   __syncthreads();

      if(x==0){
        nr=0.0;
        for(j=0;j<N;j++){
        nr+=Temp[j][k]*Temp[j][k];
        }
        }
   __syncthreads();
        q[x][k]=Temp[x][k]/sqrt(nr);
}

/**********Matrix multiplication AtB***************************************/

__global__ void matmultt(arr *l,arr *m, arr *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    __shared__ double p[N];

    int i;
    int k=threadIdx.x;
    n[x][y]=0;
    p[k]=0;
    p[k]=l[k][x]*m[k][y];

  __syncthreads();
    if(k==0){
     for(i=0;i<N;i++){
        n[x][y]=n[x][y]+p[i];
      }
   }
}

/**************matrix multiplication AB************/
__global__ void matmult(arr *l,arr *m, arr *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    __shared__ double p[N];
    int i;
    int k=threadIdx.x;

    n[x][y]=0;
    p[k]=0;
    p[k]=l[x][k]*m[k][y];
  __syncthreads();
    if(k==0){
       for(i=0;i<N;i++){
          n[x][y]=n[x][y]+p[i];
        }
     }
}

int main(int argc, char** argv) {
    int i,j,k,l,L=250;
    double time_spent = 0.0;
    clock_t begin = clock();
    size_t bytes = N * N * sizeof(double);

    // Allocate memory for our matrices
        arr *A, *q,*Temp,*temp,*MR,*KR,*FR,*LR,*r,*CR,*Q,*phiBt,*KK;
        hipMallocManaged(&A, bytes);
        hipMallocManaged(&q, bytes);
        hipMallocManaged(&Temp, bytes);
        hipMallocManaged(&temp, bytes);
        hipMallocManaged(&r, bytes);
        hipMallocManaged(&CR, bytes);
        hipMallocManaged(&Q, bytes);
        hipMallocManaged(&KK, bytes);
        hipMallocManaged(&phiBt, bytes);
        hipMallocManaged(&FR, bytes);
        hipMallocManaged(&LR, bytes);
        hipMallocManaged(&KR, bytes);
        hipMallocManaged(&MR, bytes);

/****************Import matries**********************/
        float K[N][N];
        float M[N][N];
        FILE *filek;
        filek=fopen("KG.txt","r");
        if(filek==NULL){
            printf("file doesnt exist");
            return 0;
        }
        while(!feof(filek)){
        printf("entered1");
        for(i=0;i<N;i++){
            for(j=0;j<N;j++){

        fscanf(filek, "%f", &K[i][j]);
        }}}

        FILE *file;
        file=fopen("MG.txt","r");
        if(file==NULL){
            printf("file doesnt exist");
            return 0;
        }
        while(!feof(file)){
        printf("entered2");
         for(i=0;i<N;i++){
         for(j=0;j<N;j++){
               fscanf(file, "%f", &M[i][j]);
         }}}

printf("K of A is:\n");
for(i = N-5; i < N; i++) {
for (j = N-5; j < N; j++) {
           printf("%f ",K[i][j]);
   } printf("\n");
   }

printf("M is:\n");
for(i = 0; i <10; i++) {
for(j = 0; j < 10; j++) {
           printf("%f ",M[i][j]);
   } printf("\n");
   }


//initialize q and Temp
      for(i = 0; i < N; i++) {
      for (j = 0; j < N; j++) {
             A[i][j] = M[i][j];
             q[i][j] = 0.0;
             Temp[i][j] = M[i][j];
            KK[i][j] = K[i][j];
        }
    }
//******************setting grid parameters*****************

 dim3 grid(N,N);

for(int x = 0;x< 2; x++) {
//*************QR decompose*********************
   printf("A for %dis:\n",x);
    for(i = 0; i < 10; i++) {
    for(j = 0; j < 10; j++) {
           printf("%f ",A[i][j]);
           } printf("\n");
            }

for(l = 0;l< L; l++) {
      printf("l is %d for x \n",l);

      for(i = 0; i < N; i++) {
      for (j = 0; j < N; j++) {
        q[i][j] = 0.0;
        MR[i][j] = 0.0;
        Temp[i][j] = A[i][j];
        CR[i][j] = A[i][j];

        }
        }
//Calculate q[0]
        double nr=0.0;
        for(j=0;j<N;j++){
        nr+=A[j][0]*A[j][0];
        }
        for(j=0;j<N;j++){
        q[j][0]=A[j][0]/sqrt(nr);
        }

for(k = 1;k< N; k++) {
      for(i = 0; i < N; i++) {
      for (j = 0; j < N; j++) {
           temp[i][j] = 0.0;
           r[0][i] = 0.0;
         }
        }
//**************kernel function for dot products**************

dotmat<<<grid,1>>>(A,Temp,q,k,temp);
hipDeviceSynchronize();

dot<<<1,N>>>(temp,r,Temp,q,k);
hipDeviceSynchronize();

submat<<<grid,1>>>(LR,Temp,q,k,r);
hipDeviceSynchronize();

Qcal<<<1,N>>>(LR,Temp,q,k);
hipDeviceSynchronize();

/*

        for(j=0;j<N;j++){
        for(i=0;i<N;i++){
        r[0][j]+=temp[i][j];
        }
        }

        for(j=0;j<k;j++){
        for(i=0;i<N;i++){
        Temp[i][k]=Temp[i][k]-r[0][j]*q[i][j];
        }
        }

double  nr=0.0;
        for(j=0;j<N;j++){
        nr+=Temp[j][k]*Temp[j][k];
//      nr+=r[1][j];
=0;j<N;j++){
        q[j][k]=Temp[j][k]/sqrt(nr);
        }
*/
}

//****************QRD ends****************
//*************New A cal******************
      matmultt<<<grid, N>>>(q, A, MR);
      hipDeviceSynchronize();

      matmult<<<grid,N>>>(MR, q, A);
      hipDeviceSynchronize();

//***************New Q cal*****************
    if(l<1){
        for(i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
           Q[i][j] =q[i][j];
        }
        }
    }
    if(l>0){
        matmult<<<grid, N>>>(Q, q, KR);
        hipDeviceSynchronize();
        for(i = 0; i < N; i++) {
        for (j = 0; j < N; j++){
          Q[i][j] =KR[i][j];
        }
        }
    }
//*****************************************

} //l loop

if(x<1){
//*******Cal phiBt**********
for(i = 0; i < N; i++) {
for (j = 0; j < N; j++) {
        LR[i][j] =0.00;
        if(i==j){
        LR[i][i] =1/sqrt(A[i][i]);
        }
     } }

matmult<<<grid,N>>>(Q, LR,phiBt);
hipDeviceSynchronize();

//***********************transformed K******************
matmultt<<<grid, N>>>(phiBt,KK,FR);
hipDeviceSynchronize();

matmult<<<grid, N>>>(FR, phiBt,A);
hipDeviceSynchronize();

}

} //xloop
//**************Eigen vectors********
matmult<<<grid, N>>>(phiBt, Q,CR);
hipDeviceSynchronize();

printf("Eigenvalues of A are\n");
for(i = N-5; i < N; i++) {
for (j =N-5; j < N; j++) {
           printf("%f ",A[i][j]);
   } printf("\n");
   }

printf("Eigenvectors of A is:\n");
for(i = N-5; i < N; i++) {
for (j = N-5; j < N; j++) {
           printf("%f ",CR[i][j]);
   } printf("\n");
   }


/*#####implementation of mdm#####*/
        int h,modes=10;
        double w=20000.0,dt=0.00001; //frequency
        double phi[N][modes], f[N],fmat[modes],tmat[modes][80],wr[modes],u[N][80];

        //*********setting EV and ev and f ************
        for(i=0;i<modes;i++){
        for(j=0;j<N;j++){
        phi[j][i]=CR[j][N-i-1];
        f[j]=0.0;
        }
        wr[i]=sqrt(A[N-1-i][N-1-i]);
        }
        f[89]=5*pow(10,10);
        f[440]=5*pow(10,10);

        //************cal fmat******
       for(h=0;h<modes;h++){
        fmat[h]=0;
        for(k=0;k<N;k++){
        fmat[h]+=phi[k][h]*f[k];
        }
        }
        //******q cal********
        for(i=0;i<modes;i++){
        for(j=0;j<80;j++){
          tmat[i][j]=fmat[i]*(1/(pow(wr[i],2)-pow(w,2)))*(sin(w*j*dt)-((w/wr[i])*sin(wr[i]*j*dt)));
        }
        }
        //**********u cal*********
        for(h=0;h<N;h++){
                for(j=0;j<80;j++){
                u[h][j]=0.0;
                        for(k=0;k<modes;k++){
                        u[h][j]+=phi[h][k]*tmat[k][j];
                        }
        }
        }
        printf("response\n");
        for(h=0;h<80;h++){
        printf("%8.8f ",u[89][h]);
        }

    clock_t end = clock();
    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Time elpased is %f min\n\n", time_spent/60);

return 0;
}

